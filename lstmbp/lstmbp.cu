/////////////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////////////////////////
//		Author:    Yang Song
//		File:      CUDA implementation of LSTM model including both feed-forward and back-propagation 
///////////////////////////////////////////////////////////////////////////////////////// 


#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <time.h>
#include <string>
#include <fstream>
#include <iostream>
#include <algorithm>
#include <sstream>
#include <assert.h>
using namespace std;

#define input_0 0.98
#define input_1 0.88
#define alpha 0.1

extern "C"
void computeGold(float*, const float*, const float*, unsigned int, unsigned int, unsigned int);
void NeuralNetwork();

unsigned g_verbose;
unsigned NUM;

/////////////////////////////////////////////////////////////////////////////////////////
// Program main
/////////////////////////////////////////////////////////////////////////////////////////
int main(int argc, char** argv)
{
	int i, commandline_error;
	commandline_error = 0;
	g_verbose = 0;
	if (argc >= 2) {
		NUM = atoi(argv[1]);
		for (i=2; i < argc;i++) {
			if (argv[i][0] == '-') {
				switch (argv[i][1]) {
				case 'v': g_verbose = 1;
					break;
				default: commandline_error=1;
				}
			}
			else commandline_error=1;
		}
	} else commandline_error=1;

	if (commandline_error || !NUM) {
		printf("Usage: ./LSTM <NUM> [-v]\n");
		printf("where NUM is the number of images to process in parallel (up to 10000 for the t10k-images-idx3-ubyte database file) and -v is used to display approximately what each image looks like.\n");
		return 1;
	}


	NeuralNetwork();
	printf("success\n");
       
    //CUT_EXIT(argc, argv);
}

void InitHostMem(double *w_i, double *u_i, double *b_i, double *w_f, double *u_f, double *b_f, double *w_c, double *u_c, double *b_c, double *w_o, double *u_o, double *b_o, double *weight, double *bias)
{
	// Input Gate Weights and Bias
	FILE * pFile1 = fopen ("data/w_i.txt","rb");
	if (pFile1 != NULL)
	{
		//printf("File Opened\n");
		char s[1000000] = "";
		fread(s,sizeof(s),1,pFile1);
		//printf("Reading Done\n");
		long int index = 0, i = 0;
		char delim[2];
		delim[0] = '\n';
    		delim[1] = 0;
		char* temp_string = strtok(s, delim);
		while(temp_string != NULL)
		{ 	
			double temp_num = atof(temp_string);
			w_i[i] = temp_num;
			i++;
			index++;
			if(i==100)
			{
				//printf("Breaking\n");
				break;
			}
			temp_string = strtok(NULL, delim);
		}
		fclose (pFile1);
	}	
	
	if (!pFile1)
	{
		printf("FAIL! INPUT WEIGHTS NOT FOUND!\n");
		exit(1);
	}
	FILE * pFile2 = fopen ("data/b_i.txt","rb");
	if (pFile2 != NULL)
	{
		//printf("File Opened\n");
		char s[1000000] = "";
		fread(s,sizeof(s),1,pFile2);
		//printf("Reading Done\n");
		long int index = 0, i = 0;
		char delim[2];
		delim[0] = '\n';
    		delim[1] = 0;
		char* temp_string = strtok(s, delim);
		while(temp_string != NULL)
		{ 	
			double temp_num = atof(temp_string);
			b_i[i] = temp_num;
			i++;
			index++;
			if(i==100)
			{
				//printf("Breaking\n");
				break;
			}
			temp_string = strtok(NULL, delim);
		}
		fclose (pFile2);
	}	
	//cout<<"Input gate reading completed, "<<b_i[99]<<endl;
	if (!pFile2)
	{
		printf("FAIL! INPUT WEIGHTS NOT FOUND!\n");
		exit(1);
	}
	// Forget gate Weights and Bias
	FILE * pFile3 = fopen ("data/w_f.txt","rb");
	if (pFile3 != NULL)
	{
		//printf("File Opened\n");
		char s[1000000] = "";
		fread(s,sizeof(s),1,pFile3);
		//printf("Reading Done\n");
		long int index = 0, i = 0;
		char delim[2];
		delim[0] = '\n';
    		delim[1] = 0;
		char* temp_string = strtok(s, delim);
		while(temp_string != NULL)
		{ 	
			double temp_num = atof(temp_string);
			w_f[i] = temp_num;
			i++;
			index++;
			if(i==100)
			{
				//printf("Breaking\n");
				break;
			}
			temp_string = strtok(NULL, delim);
		}
		fclose (pFile3);
	}	
	
	if (!pFile3)
	{
		printf("FAIL! INPUT WEIGHTS NOT FOUND!\n");
		exit(1);
	}
	FILE * pFile4 = fopen ("data/b_f.txt","rb");
	if (pFile4 != NULL)
	{
		//printf("File Opened\n");
		char s[1000000] = "";
		fread(s,sizeof(s),1,pFile4);
		//printf("Reading Done\n");
		long int index = 0, i = 0;
		char delim[2];
		delim[0] = '\n';
    		delim[1] = 0;
		char* temp_string = strtok(s, delim);
		while(temp_string != NULL)
		{ 	
			double temp_num = atof(temp_string);
			b_f[i] = temp_num;
			i++;
			index++;
			if(i==100)
			{
				//printf("Breaking\n");
				break;
			}
			temp_string = strtok(NULL, delim);
		}
		fclose (pFile4);
	}	
	//cout<<"Forget gate reading completed, "<<b_f[99]<<endl;
	if (!pFile4)
	{
		printf("FAIL! INPUT WEIGHTS NOT FOUND!\n");
		exit(1);
	}
	// Cell gate Weights and Bias
	FILE * pFile5 = fopen ("data/w_c.txt","rb");
	if (pFile5 != NULL)
	{
		//printf("File Opened\n");
		char s[1000000] = "";
		fread(s,sizeof(s),1,pFile5);
		//printf("Reading Done\n");
		long int index = 0, i = 0;
		char delim[2];
		delim[0] = '\n';
    		delim[1] = 0;
		char* temp_string = strtok(s, delim);
		while(temp_string != NULL)
		{ 	
			double temp_num = atof(temp_string);
			w_c[i] = temp_num;
			i++;
			index++;
			if(i==100)
			{
				//printf("Breaking\n");
				break;
			}
			temp_string = strtok(NULL, delim);
		}
		fclose (pFile5);
	}	
	
	if (!pFile5)
	{
		printf("FAIL! INPUT WEIGHTS NOT FOUND!\n");
		exit(1);
	}
	FILE * pFile6 = fopen ("data/b_c.txt","rb");
	if (pFile6 != NULL)
	{
		//printf("File Opened\n");
		char s[1000000] = "";
		fread(s,sizeof(s),1,pFile6);
		//printf("Reading Done\n");
		long int index = 0, i = 0;
		char delim[2];
		delim[0] = '\n';
    		delim[1] = 0;
		char* temp_string = strtok(s, delim);
		while(temp_string != NULL)
		{ 	
			double temp_num = atof(temp_string);
			b_c[i] = temp_num;
			i++;
			index++;
			if(i==100)
			{
				//printf("Breaking\n");
				break;
			}
			temp_string = strtok(NULL, delim);
		}
		fclose (pFile6);
	}	
	//cout<<"Forget gate reading completed, "<<b_c[99]<<endl;
	if (!pFile6)
	{
		printf("FAIL! INPUT WEIGHTS NOT FOUND!\n");
		exit(1);
	}
	// Output gate Weights and Bias
	FILE * pFile7 = fopen ("data/w_o.txt","rb");
	if (pFile7 != NULL)
	{
		//printf("File Opened\n");
		char s[1000000] = "";
		fread(s,sizeof(s),1,pFile7);
		//printf("Reading Done\n");
		long int index = 0, i = 0;
		char delim[2];
		delim[0] = '\n';
    		delim[1] = 0;
		char* temp_string = strtok(s, delim);
		while(temp_string != NULL)
		{ 	
			double temp_num = atof(temp_string);
			w_o[i] = temp_num;
			i++;
			index++;
			if(i==100)
			{
				//printf("Breaking\n");
				break;
			}
			temp_string = strtok(NULL, delim);
		}
		fclose (pFile7);
	}	
	
	if (!pFile7)
	{
		printf("FAIL! INPUT WEIGHTS NOT FOUND!\n");
		exit(1);
	}
	FILE * pFile8 = fopen ("data/b_o.txt","rb");
	if (pFile8 != NULL)
	{
		//printf("File Opened\n");
		char s[1000000] = "";
		fread(s,sizeof(s),1,pFile8);
		//printf("Reading Done\n");
		long int index = 0, i = 0;
		char delim[2];
		delim[0] = '\n';
    		delim[1] = 0;
		char* temp_string = strtok(s, delim);
		while(temp_string != NULL)
		{ 	
			double temp_num = atof(temp_string);
			b_o[i] = temp_num;
			i++;
			index++;
			if(i==100)
			{
				//printf("Breaking\n");
				break;
			}
			temp_string = strtok(NULL, delim);
		}
		fclose (pFile8);
	}	
	//cout<<"Output gate reading completed, "<<b_o[99]<<endl;
	if (!pFile8)
	{
		printf("FAIL! INPUT WEIGHTS NOT FOUND!\n");
		exit(1);
	}
	FILE * pFile9 = fopen ("data/W.txt","rb");
	if (pFile9 != NULL)
	{
		//printf("File Opened\n");
		char s[1000000] = "";
		fread(s,sizeof(s),1,pFile9);
		//printf("Reading Done\n");
		long int index = 0, i = 0;
		char delim[2];
		delim[0] = '\n';
    		delim[1] = 0;
		char* temp_string = strtok(s, delim);
		while(temp_string != NULL)
		{ 	
			double temp_num = atof(temp_string);
			weight[i] = temp_num;
			i++;
			index++;
			if(i==101)
			{
				//printf("Breaking\n");
				break;
			}
			temp_string = strtok(NULL, delim);
		}
		fclose (pFile9);
	}
	*bias = weight[100];	
	//cout<<"Fully Connected layer reading completed, "<<*bias<<endl;
	if (!pFile9)
	{
		printf("FAIL! INPUT WEIGHTS NOT FOUND!\n");
		exit(1);
	}
	//Recurrent weights of output gate
	FILE * pFile10 = fopen ("data/u_o.txt","rb");
	if (pFile10 != NULL)
	{
		//printf("File Opened\n");
		char s[1000000] = "";
		fread(s,sizeof(s),1,pFile10);
		//printf("Reading Done\n");
		long int index = 0, i = 0;
		char delim[2];
		delim[0] = '\n';
    		delim[1] = 0;
		char* temp_string = strtok(s, delim);
		while(temp_string != NULL)
		{ 	
			double temp_num = atof(temp_string);
			u_o[i] = temp_num;
			i++;
			index++;
			if(i==10000)
			{
				//printf("Breaking\n");
				break;
			}
			temp_string = strtok(NULL, delim);
		}
		fclose (pFile10);
	}	
	
	if (!pFile10)
	{
		printf("FAIL! INPUT WEIGHTS NOT FOUND!\n");
		exit(1);
	}
	//Recurrent weights of input gate
	FILE * pFile11 = fopen ("data/u_i.txt","rb");
	if (pFile11 != NULL)
	{
		//printf("File Opened\n");
		char s[1000000] = "";
		fread(s,sizeof(s),1,pFile11);
		//printf("Reading Done\n");
		long int index = 0, i = 0;
		char delim[2];
		delim[0] = '\n';
    		delim[1] = 0;
		char* temp_string = strtok(s, delim);
		while(temp_string != NULL)
		{ 	
			double temp_num = atof(temp_string);
			u_i[i] = temp_num;
			i++;
			index++;
			if(i==10000)
			{
				//printf("Breaking\n");
				break;
			}
			temp_string = strtok(NULL, delim);
		}
		fclose (pFile11);
	}	
	
	if (!pFile11)
	{
		printf("FAIL! INPUT WEIGHTS NOT FOUND!\n");
		exit(1);
	}
	//Recurrent weights of cell gate
	FILE * pFile12 = fopen ("data/u_c.txt","rb");
	if (pFile12 != NULL)
	{
		//printf("File Opened\n");
		char s[1000000] = "";
		fread(s,sizeof(s),1,pFile12);
		//printf("Reading Done\n");
		long int index = 0, i = 0;
		char delim[2];
		delim[0] = '\n';
    		delim[1] = 0;
		char* temp_string = strtok(s, delim);
		while(temp_string != NULL)
		{ 	
			double temp_num = atof(temp_string);
			u_c[i] = temp_num;
			i++;
			index++;
			if(i==10000)
			{
				//printf("Breaking\n");
				break;
			}
			temp_string = strtok(NULL, delim);
		}
		fclose (pFile12);
	}	
	
	if (!pFile12)
	{
		printf("FAIL! INPUT WEIGHTS NOT FOUND!\n");
		exit(1);
	}
	//Recurrent weights of forget gate
	FILE * pFile13 = fopen ("data/u_f.txt","rb");
	if (pFile13 != NULL)
	{
		//printf("File Opened\n");
		char s[1000000] = "";
		fread(s,sizeof(s),1,pFile13);
		//printf("Reading Done\n");
		long int index = 0, i = 0;
		char delim[2];
		delim[0] = '\n';
    		delim[1] = 0;
		char* temp_string = strtok(s, delim);
		while(temp_string != NULL)
		{ 	
			double temp_num = atof(temp_string);
			u_f[i] = temp_num;
			i++;
			index++;
			if(i==10000)
			{
				//printf("Breaking\n");
				break;
			}
			temp_string = strtok(NULL, delim);
		}
		fclose (pFile13);
	}	
	
	if (!pFile13)
	{
		printf("FAIL! INPUT WEIGHTS NOT FOUND!\n");
		exit(1);
	}
}

__device__ double hard_sigmoid(double x)
{
	if(x<-2.5)
		return 0;
	else
	{
		if(x>2.5)
			return 1;
		else
			return (0.2*x + 0.5);
	}
}

__device__ double dsigmoid(double y)
{
	return  y * (1.0 - y);
}

__device__ double dtanh( double y)
{
	y = tanh(y);
	return 1.0 - y * y;
	
}
 
__global__ void ExecuteLSTM( double result_0, double result_1, double *w_i, double *u_i, double *b_i, double *w_f, double *u_f, double *b_f, double *w_c, double *u_c, double *b_c, double *w_o, double *u_o, double *b_o, double *weight, double *bias, double *LSTM_results, double *at_0, double *at_1, double *it_0, double *it_1, double *ft_0, double *ft_1, double *ot_0, double *ot_1, double *statet_0, double *statet_1, double *u_0, double* u_1)
{
	int x = threadIdx.x;
	double c = 0;
	double i[2];
	double temp_ua = 0, temp_ui = 0, temp_uf = 0, temp_uo = 0;
	//int index = threadIdx.x*2;
	i[0] = input_0;
	i[1] = input_1;


	at_0[x] = tanh(i[0]*w_c[x] + b_c[x]);
	it_0[x] = hard_sigmoid(i[0]*w_i[x] + b_i[x]);
        ft_0[x] = hard_sigmoid(i[0]*w_f[x] + b_f[x]);
	ot_0[x] = hard_sigmoid(i[0]*w_o[x] + b_o[x]);
	statet_0[x] = at_0[x]*it_0[x] + ft_0[x]*c;
	c = statet_0[x];
	u_0[x] = ot_0[x]*tanh(statet_0[x]);
	
	__syncthreads();
	if(x==0)
	{
		double result_0 = *bias;
		for(int i=0; i<100; i++)
		{
			result_0 += weight[i]*u_0[i];
		}
		//printf("The result for i=0 is %f\n",result);
	}
	__syncthreads();
	for(int i=0; i<100; i++)
	{
		temp_ua += u_0[i] * u_c[i*100 + x];
		temp_ui += u_0[i] * u_i[i*100 + x];
		temp_uf += u_0[i] * u_f[i*100 + x];
		temp_uo += u_0[i] * u_o[i*100 + x]; 	
	}
	
	at_1[x] = tanh(i[1]*w_c[x] + temp_ua + b_c[x]);//新记忆
	it_1[x] = hard_sigmoid(i[1]*w_i[x] + temp_ui + b_i[x]);
	ft_1[x] = hard_sigmoid(i[1]*w_f[x] + temp_uf + b_f[x]);
	ot_1[x] = hard_sigmoid(i[1]*w_o[x] + temp_uo + b_o[x]);
	
	statet_1[x]= at_1[x]*it_1[x] + ft_1[x]*c; //最终记忆
	u_1[x] = ot_1[x]*tanh(statet_1[x]);
	//隐藏层输出
	__syncthreads();
	if(x==0)
	{
		double result_1 = *bias;
		for(int i=0; i<100; i++)
		{
			result_1 += weight[i]*u_1[i];
			//printf("u[i]=\n",u_1[i]);
		}
		//printf("The result is %f\n",result_1);
		
	}
}

__global__ void bpLSTM( double result_0, double result_1, double *w_i, double *u_i, double *b_i, double *w_f, double *u_f, double *b_f, double *w_c, double *u_c, double *b_c, double *w_o, double *u_o, double *b_o, double *weight, double *bias, double *LSTM_results, double *at_0, double *at_1, double *it_0, double *it_1, double *ft_0, double *ft_1, double *ot_0, double *ot_1, double *statet_0, double *statet_1, double *u_0, double *u_1)
{
       int x = threadIdx.x;
       double i[2];
       i[0] = input_0; 
       i[1] = input_1;
       __shared__ double u_delta[100];
       __shared__ double O_delta[100];
       __shared__ double I_delta[100];
       __shared__ double F_delta[100];
       __shared__ double A_delta[100];
       __shared__ double state_delta[100];
       __shared__ double O_future_delta[100];
       __shared__ double I_future_delta[100];
       __shared__ double F_future_delta[100];
       __shared__ double A_future_delta[100];
       __shared__ double state_future_delta[100];
       __shared__ double forget_gate_future[100];

       __shared__ double u_pre[100];
       __shared__ double state_pre[100];
      
  
       O_future_delta[x] = 0.0;
       I_future_delta[x] = 0.0;
       F_future_delta[x] = 0.0;
       A_future_delta[x] = 0.0;
       state_future_delta[x] = 0.0;
       forget_gate_future[x] = 0.0;
       u_pre[x] = 0.0;
       state_pre[x]= 0.0;
     
       __syncthreads();
       u_delta[x] = 0;
       weight[x] += alpha * (input_1 - result_1) * dsigmoid(result_1) * u_1[x];
       u_delta[x] += (input_1 - result_1) * dsigmoid(result_1) * weight[x];
       for (int k = 0; k < 100; k++)
       {
    	    u_delta[x] += I_future_delta[k] * u_i[x*100+k];
    	    u_delta[x] += F_future_delta[k] * u_f[x*100+k];
    	    u_delta[x] += O_future_delta[k] * u_o[x*100+k];
    	    u_delta[x] += A_future_delta[k] * u_c[x*100+k];
       }
       O_delta[x] = u_delta[x] * tanh(statet_1[x]) * dsigmoid(ot_1[x]);
       state_delta[x] = u_delta[x] * ot_1[x] * dtanh(statet_1[x]) + state_future_delta[x] *  forget_gate_future[x];
       F_delta[x] = state_delta[x] * statet_0[x] * dsigmoid(ft_1[x]);
       I_delta[x] = state_delta[x] * at_1[x] * dsigmoid(it_1[x]);
       A_delta[x] = state_delta[x] * it_1[x] * dtanh(at_1[x]);


       for(int k = 0; k < 100; k++)
       {
    	    u_i[k*100+x] += alpha * I_delta[x] * u_0[k];
    	    u_f[k*100+x] += alpha * F_delta[x] * u_0[k];
    	    u_o[k*100+x] += alpha * O_delta[x] * u_0[k];
    	    u_c[k*100+x] += alpha * A_delta[x] * u_0[k];
       }

       w_i[x] += alpha * I_delta[x] * i[1];
       w_f[x] += alpha * F_delta[x] * i[1];
       w_o[x] += alpha * O_delta[x] * i[1];
       w_c[x] += alpha * A_delta[x] * i[1];

       __syncthreads();
    
      // double *u_pre = new double[100];
       //double *state_pre = new double[100];
      // for (int j = 0; j < 100; j++) //第一个隐藏层输出，相当于h_(t-1)
      // {
       //     u_pre[j] = 0.0;
	//    state_pre[j] = 0.0;
      // }
       u_delta[x] = 0;
       weight[x] += alpha * (input_0 - result_0) * dsigmoid(result_0) * u_0[x];
       u_delta[x] += (input_0 - result_0) * dsigmoid(result_0) * weight[x];
       for (int  k = 0; k < 100; k++)
       {
    	    u_delta[x] += I_delta[k] * u_i[x*100+k];
    	    u_delta[x] += F_delta[k] * u_f[x*100+k];
    	    u_delta[x] += O_delta[k] * u_o[x*100+k];
    	    u_delta[x] += A_delta[k] * u_c[x*100+k];
       }
       O_delta[x] = u_delta[x] * tanh(statet_0[x]) * dsigmoid(ot_0[x]);
       state_delta[x] = u_delta[x] * ot_0[x] * dtanh(statet_0[x]) +  statet_1[x] * ft_1[x];
       F_delta[x] = u_delta[x] * state_pre[x] * dsigmoid(ft_0[x]);
       I_delta[x] = u_delta[x] * at_0[x] * dsigmoid(it_0[x]);
       A_delta[x] = u_delta[x] * it_0[x] * dtanh(at_0[x]);
       for (int k = 0; k < 100; k++)
       {
    	    u_i[k*100+x] += alpha * I_delta[x] * u_pre[k];
    	    u_f[k*100+x] += alpha * F_delta[x] * u_pre[k];
    	    u_o[k*100+x] += alpha * O_delta[x] * u_pre[k];
    	    u_c[k*100+x] += alpha * A_delta[x] * u_pre[k];
       }
       w_i[x] += alpha * I_delta[x] * i[0];
       w_f[x] += alpha * F_delta[x] * i[0];
       w_o[x] += alpha * O_delta[x] * i[0];
       w_c[x] += alpha * A_delta[x] * i[0];
       __syncthreads();

}




void NeuralNetwork()
{
	hipError_t err;
	//cudaEvent_t start, stop;
	double *w_i = (double*) malloc (100 * NUM * sizeof(double));
	double *u_i = (double*) malloc (10000 * NUM * sizeof(double));
	double *b_i = (double*) malloc (100 * NUM * sizeof(double));
	double *w_f = (double*) malloc (100 * NUM * sizeof(double));
	double *u_f = (double*) malloc (10000 * NUM * sizeof(double));
	double *b_f = (double*) malloc (100 * NUM * sizeof(double));
	double *w_c = (double*) malloc (100 * NUM * sizeof(double));
	double *u_c = (double*) malloc (10000 * NUM * sizeof(double));
	double *b_c = (double*) malloc (100 * NUM * sizeof(double));
	double *w_o = (double*) malloc (100 * NUM * sizeof(double));
	double *u_o = (double*) malloc (10000 * NUM * sizeof(double));
	double *b_o = (double*) malloc (100 * NUM * sizeof(double));
	double *weights = (double*) malloc (101 * NUM * sizeof(double));
	double *bias = (double*) malloc (sizeof(double));
        double *at_0 = (double*) malloc (100 * NUM * sizeof(double));
        double *it_0 = (double*) malloc (100 * NUM * sizeof(double));
        double *ft_0 = (double*) malloc (100 * NUM * sizeof(double));
        double *ot_0 = (double*) malloc (100 * NUM * sizeof(double));
        double *statet_0 = (double*) malloc (100 * NUM * sizeof(double));
        double *u_0 = (double*) malloc (100 * NUM * sizeof(double));
        double *at_1 = (double*) malloc (100 * NUM * sizeof(double));
        double *it_1 = (double*) malloc (100 * NUM * sizeof(double));
        double *ft_1 = (double*) malloc (100 * NUM * sizeof(double));
        double *ot_1 = (double*) malloc (100 * NUM * sizeof(double));
        double *statet_1 = (double*) malloc (100 * NUM * sizeof(double));
        double *u_1 = (double*) malloc (100 * NUM * sizeof(double));


    




	InitHostMem(w_i, u_i, b_i, w_f, u_f, b_f, w_c, u_c, b_c, w_o, u_o, b_o, weights, bias);
	//cout<<*bias<<endl;
	double *w_i_device;
	err = hipMalloc((void**) &w_i_device, 100* NUM * sizeof(double));
	if (err != hipSuccess)
        {
        	fprintf(stderr, "Failed to allocate device data (error code %s)!\n", hipGetErrorString(err));
        	exit(EXIT_FAILURE);
        }
	double *u_i_device;
	err = hipMalloc((void**) &u_i_device, 10000* NUM * sizeof(double));
	if (err != hipSuccess)
        {
        	fprintf(stderr, "Failed to allocate device data (error code %s)!\n", hipGetErrorString(err));
        	exit(EXIT_FAILURE);
        }
	double *b_i_device;
	err = hipMalloc((void**) &b_i_device, 100* NUM * sizeof(double));
	if (err != hipSuccess)
        {
        	fprintf(stderr, "Failed to allocate device data (error code %s)!\n", hipGetErrorString(err));
        	exit(EXIT_FAILURE);
        }
	double *w_f_device;
	err = hipMalloc((void**) &w_f_device, 100* NUM * sizeof(double));
	if (err != hipSuccess)
        {
        	fprintf(stderr, "Failed to allocate device data (error code %s)!\n", hipGetErrorString(err));
        	exit(EXIT_FAILURE);
        }
	double *u_f_device;
	err = hipMalloc((void**) &u_f_device, 10000* NUM * sizeof(double));
	if (err != hipSuccess)
        {
        	fprintf(stderr, "Failed to allocate device data (error code %s)!\n", hipGetErrorString(err));
        	exit(EXIT_FAILURE);
        }
	double *b_f_device;
	err = hipMalloc((void**) &b_f_device, 100* NUM * sizeof(double));
	if (err != hipSuccess)
        {
        	fprintf(stderr, "Failed to allocate device data (error code %s)!\n", hipGetErrorString(err));
        	exit(EXIT_FAILURE);
        }
	double *w_c_device;
	err = hipMalloc((void**) &w_c_device, 100* NUM * sizeof(double));
	if (err != hipSuccess)
        {
        	fprintf(stderr, "Failed to allocate device data (error code %s)!\n", hipGetErrorString(err));
        	exit(EXIT_FAILURE);
        }
	double *u_c_device;
	err = hipMalloc((void**) &u_c_device, 10000* NUM * sizeof(double));
	if (err != hipSuccess)
        {
        	fprintf(stderr, "Failed to allocate device data (error code %s)!\n", hipGetErrorString(err));
        	exit(EXIT_FAILURE);
        }
	double *b_c_device;
	err = hipMalloc((void**) &b_c_device, 100* NUM * sizeof(double));
	if (err != hipSuccess)
        {
        	fprintf(stderr, "Failed to allocate device data (error code %s)!\n", hipGetErrorString(err));
        	exit(EXIT_FAILURE);
        }
	double *w_o_device;
	err = hipMalloc((void**) &w_o_device, 100* NUM * sizeof(double));
	if (err != hipSuccess)
        {
        	fprintf(stderr, "Failed to allocate device data (error code %s)!\n", hipGetErrorString(err));
        	exit(EXIT_FAILURE);
        }
	double *u_o_device;
	err = hipMalloc((void**) &u_o_device, 10000* NUM * sizeof(double));
	if (err != hipSuccess)
        {
        	fprintf(stderr, "Failed to allocate device data (error code %s)!\n", hipGetErrorString(err));
        	exit(EXIT_FAILURE);
        }
	double *b_o_device;
	err = hipMalloc((void**) &b_o_device, 100* NUM * sizeof(double));
	if (err != hipSuccess)
        {
        	fprintf(stderr, "Failed to allocate device data (error code %s)!\n", hipGetErrorString(err));
        	exit(EXIT_FAILURE);
        }
	double *weights_device;
	err = hipMalloc((void**) &weights_device, 100* NUM * sizeof(double));
	if (err != hipSuccess)
        {
        	fprintf(stderr, "Failed to allocate device data (error code %s)!\n", hipGetErrorString(err));
        	exit(EXIT_FAILURE);
        }
	double *bias_device;
	err = hipMalloc((void**) &bias_device, 1* NUM * sizeof(double));
	if (err != hipSuccess)
        {
        	fprintf(stderr, "Failed to allocate device data (error code %s)!\n", hipGetErrorString(err));
        	exit(EXIT_FAILURE);
        }
	double *LSTM_results;
	err = hipMalloc((void**) &LSTM_results, 100* NUM * sizeof(double));
	if (err != hipSuccess)
        {
        	fprintf(stderr, "Failed to allocate device data (error code %s)!\n", hipGetErrorString(err));
        	exit(EXIT_FAILURE);
        }
	//printf("Malloc completed\n");
	//Start Memory Copy
	err = hipMemcpy(w_i_device, w_i, sizeof(double)*100*NUM, hipMemcpyHostToDevice);
	if (err != hipSuccess)
        {
        	fprintf(stderr, "Failed to copy device data 1(error code %s)!\n", hipGetErrorString(err));
        	exit(EXIT_FAILURE);
        }
	err = hipMemcpy(u_i_device, u_i, sizeof(double)*10000*NUM, hipMemcpyHostToDevice);
	if (err != hipSuccess)
        {
        	fprintf(stderr, "Failed to copy device data 2(error code %s)!\n", hipGetErrorString(err));
        	exit(EXIT_FAILURE);
        }
	err = hipMemcpy(b_i_device, b_i, sizeof(double)*100*NUM, hipMemcpyHostToDevice);
	if (err != hipSuccess)
        {
        	fprintf(stderr, "Failed to copy device data 3(error code %s)!\n", hipGetErrorString(err));
        	exit(EXIT_FAILURE);
        }
	err = hipMemcpy(w_f_device, w_f, sizeof(double)*100*NUM, hipMemcpyHostToDevice);
	if (err != hipSuccess)
        {
        	fprintf(stderr, "Failed to copy device data 4(error code %s)!\n", hipGetErrorString(err));
        	exit(EXIT_FAILURE);
        }
	err = hipMemcpy(u_f_device, u_f, sizeof(double)*10000*NUM, hipMemcpyHostToDevice);
	if (err != hipSuccess)
        {
        	fprintf(stderr, "Failed to copy device data 5(error code %s)!\n", hipGetErrorString(err));
        	exit(EXIT_FAILURE);
        }
	err = hipMemcpy(b_f_device, b_f, sizeof(double)*100*NUM, hipMemcpyHostToDevice);
	if (err != hipSuccess)
        {
        	fprintf(stderr, "Failed to copy device data 6(error code %s)!\n", hipGetErrorString(err));
        	exit(EXIT_FAILURE);
        }
	err = hipMemcpy(w_c_device, w_c, sizeof(double)*100*NUM, hipMemcpyHostToDevice);
	if (err != hipSuccess)
        {
        	fprintf(stderr, "Failed to copy device data 7(error code %s)!\n", hipGetErrorString(err));
        	exit(EXIT_FAILURE);
        }
	err = hipMemcpy(u_c_device, u_c, sizeof(double)*10000*NUM, hipMemcpyHostToDevice);
	if (err != hipSuccess)
        {
        	fprintf(stderr, "Failed to copy device data 8(error code %s)!\n", hipGetErrorString(err));
        	exit(EXIT_FAILURE);
        }
	err = hipMemcpy(b_c_device, b_c, sizeof(double)*100*NUM, hipMemcpyHostToDevice);
	if (err != hipSuccess)
        {
        	fprintf(stderr, "Failed to copy device data 9(error code %s)!\n", hipGetErrorString(err));
        	exit(EXIT_FAILURE);
        }
	err = hipMemcpy(w_o_device, w_o, sizeof(double)*100*NUM, hipMemcpyHostToDevice);
	if (err != hipSuccess)
        {
        	fprintf(stderr, "Failed to copy device data (error code %s)!\n", hipGetErrorString(err));
        	exit(EXIT_FAILURE);
        }
	err = hipMemcpy(u_o_device, u_o, sizeof(double)*10000*NUM, hipMemcpyHostToDevice);
	if (err != hipSuccess)
        {
        	fprintf(stderr, "Failed to copy device data (error code %s)!\n", hipGetErrorString(err));
        	exit(EXIT_FAILURE);
        }
	err = hipMemcpy(b_o_device, b_o, sizeof(double)*100*NUM, hipMemcpyHostToDevice);
	if (err != hipSuccess)
        {
        	fprintf(stderr, "Failed to copy device data (error code %s)!\n", hipGetErrorString(err));
        	exit(EXIT_FAILURE);
        }
	err = hipMemcpy(weights_device, weights, sizeof(double)*100*NUM, hipMemcpyHostToDevice);
	if (err != hipSuccess)
        {
        	fprintf(stderr, "Failed to copy device data (error code %s)!\n", hipGetErrorString(err));
        	exit(EXIT_FAILURE);
        }
	err = hipMemcpy(bias_device, bias, sizeof(double)*1*NUM, hipMemcpyHostToDevice);
	if (err != hipSuccess)
        {
        	fprintf(stderr, "Failed to copy device bias data (error code %s)!\n", hipGetErrorString(err));
        	exit(EXIT_FAILURE);
        }
    



        double *at_0_device;
	err = hipMalloc((void**) &at_0_device, 100* NUM * sizeof(double));
	if (err != hipSuccess)
        {
        	fprintf(stderr, "Failed to allocate device data (error code %s)!\n", hipGetErrorString(err));
        	exit(EXIT_FAILURE);
        }
        double *it_0_device;
	err = hipMalloc((void**) &it_0_device, 100* NUM * sizeof(double));
	if (err != hipSuccess)
        {
        	fprintf(stderr, "Failed to allocate device data (error code %s)!\n", hipGetErrorString(err));
        	exit(EXIT_FAILURE);
        }
        double *ft_0_device;
	err = hipMalloc((void**) &ft_0_device, 100* NUM * sizeof(double));
	if (err != hipSuccess)
        {
        	fprintf(stderr, "Failed to allocate device data (error code %s)!\n", hipGetErrorString(err));
        	exit(EXIT_FAILURE);

        }
        double *ot_0_device;
	err = hipMalloc((void**) &ot_0_device, 100* NUM * sizeof(double));
	if (err != hipSuccess)
        {
        	fprintf(stderr, "Failed to allocate device data (error code %s)!\n", hipGetErrorString(err));
        	exit(EXIT_FAILURE);
        }
        double *statet_0_device;
	err = hipMalloc((void**) &statet_0_device, 100* NUM * sizeof(double));
	if (err != hipSuccess)
        {
        	fprintf(stderr, "Failed to allocate device data (error code %s)!\n", hipGetErrorString(err));
        	exit(EXIT_FAILURE);
        }
        double *u_0_device;
	err = hipMalloc((void**) &u_0_device, 100* NUM * sizeof(double));
	if (err != hipSuccess)
        {
        	fprintf(stderr, "Failed to allocate device data (error code %s)!\n", hipGetErrorString(err));
        	exit(EXIT_FAILURE);
        }
    





        double *at_1_device;
	err = hipMalloc((void**) &at_1_device, 100* NUM * sizeof(double));
	if (err != hipSuccess)
        {
        	fprintf(stderr, "Failed to allocate device data (error code %s)!\n", hipGetErrorString(err));
        	exit(EXIT_FAILURE);
        }
        double *it_1_device;
	err = hipMalloc((void**) &it_1_device, 100* NUM * sizeof(double));
	if (err != hipSuccess)
        {
        	fprintf(stderr, "Failed to allocate device data (error code %s)!\n", hipGetErrorString(err));
        	exit(EXIT_FAILURE);
        }
        double *ft_1_device;
	err = hipMalloc((void**) &ft_1_device, 100* NUM * sizeof(double));
	if (err != hipSuccess)
        {
        	fprintf(stderr, "Failed to allocate device data (error code %s)!\n", hipGetErrorString(err));
        	exit(EXIT_FAILURE);

        }
        double *ot_1_device;
	err = hipMalloc((void**) &ot_1_device, 100* NUM * sizeof(double));
	if (err != hipSuccess)
        {
        	fprintf(stderr, "Failed to allocate device data (error code %s)!\n", hipGetErrorString(err));
        	exit(EXIT_FAILURE);
        }
        double *statet_1_device;
	err = hipMalloc((void**) &statet_1_device, 100* NUM * sizeof(double));
	if (err != hipSuccess)
        {
        	fprintf(stderr, "Failed to allocate device data (error code %s)!\n", hipGetErrorString(err));
        	exit(EXIT_FAILURE);
        }
        double *u_1_device;
	err = hipMalloc((void**) &u_1_device, 100* NUM * sizeof(double));
	if (err != hipSuccess)
        {
        	fprintf(stderr, "Failed to allocate device data (error code %s)!\n", hipGetErrorString(err));
        	exit(EXIT_FAILURE);
        }

    //printf("Malloc completed\n");
	//Start Memory Copy
	err = hipMemcpy(at_0_device, at_0, sizeof(double)*100*NUM, hipMemcpyHostToDevice);
	if (err != hipSuccess)
        {
        	fprintf(stderr, "Failed to copy device data (error code %s)!\n", hipGetErrorString(err));
        	exit(EXIT_FAILURE);
        }
	err = hipMemcpy(it_0_device, it_0, sizeof(double)*100*NUM, hipMemcpyHostToDevice);
	if (err != hipSuccess)
        {
        	fprintf(stderr, "Failed to copy device data (error code %s)!\n", hipGetErrorString(err));
        	exit(EXIT_FAILURE);
        }
        err = hipMemcpy(ft_0_device, ft_0, sizeof(double)*100*NUM, hipMemcpyHostToDevice);
	if (err != hipSuccess)
        {
        	fprintf(stderr, "Failed to copy device data (error code %s)!\n", hipGetErrorString(err));
        	exit(EXIT_FAILURE);
        }
        err = hipMemcpy(ot_0_device, ot_0, sizeof(double)*100*NUM, hipMemcpyHostToDevice);
	if (err != hipSuccess)
        {
        	fprintf(stderr, "Failed to copy device data (error code %s)!\n", hipGetErrorString(err));
        	exit(EXIT_FAILURE);
        }
        err = hipMemcpy(statet_0_device, statet_0, sizeof(double)*100*NUM, hipMemcpyHostToDevice);
	if (err != hipSuccess)
        {
        	fprintf(stderr, "Failed to copy device data (error code %s)!\n", hipGetErrorString(err));
        	exit(EXIT_FAILURE);
        }
        err = hipMemcpy(u_0_device, u_0, sizeof(double)*100*NUM, hipMemcpyHostToDevice);
	if (err != hipSuccess)
        {
        	fprintf(stderr, "Failed to copy device data (error code %s)!\n", hipGetErrorString(err));
        	exit(EXIT_FAILURE);
        }
        err = hipMemcpy(at_1_device, at_1, sizeof(double)*100*NUM, hipMemcpyHostToDevice);
	if (err != hipSuccess)
        {
        	fprintf(stderr, "Failed to copy device data (error code %s)!\n", hipGetErrorString(err));
        	exit(EXIT_FAILURE);
        }
	err = hipMemcpy(it_1_device, it_1, sizeof(double)*100*NUM, hipMemcpyHostToDevice);
	if (err != hipSuccess)
        {
        	fprintf(stderr, "Failed to copy device data (error code %s)!\n", hipGetErrorString(err));
        	exit(EXIT_FAILURE);
        }
        err = hipMemcpy(ft_1_device, ft_1, sizeof(double)*100*NUM, hipMemcpyHostToDevice);
	if (err != hipSuccess)
        {
        	fprintf(stderr, "Failed to copy device data (error code %s)!\n", hipGetErrorString(err));
        	exit(EXIT_FAILURE);
        }
        err = hipMemcpy(ot_1_device, ot_1, sizeof(double)*100*NUM, hipMemcpyHostToDevice);
	if (err != hipSuccess)
        {
        	fprintf(stderr, "Failed to copy device data (error code %s)!\n", hipGetErrorString(err));
        	exit(EXIT_FAILURE);
        }
        err = hipMemcpy(statet_1_device, statet_1, sizeof(double)*100*NUM, hipMemcpyHostToDevice);
	if (err != hipSuccess)
        {
        	fprintf(stderr, "Failed to copy device data (error code %s)!\n", hipGetErrorString(err));
        	exit(EXIT_FAILURE);
        }
        err = hipMemcpy(u_1_device, u_1, sizeof(double)*100*NUM, hipMemcpyHostToDevice);
	if (err != hipSuccess)
        {
        	fprintf(stderr, "Failed to copy device data (error code %s)!\n", hipGetErrorString(err));
        	exit(EXIT_FAILURE);
        }

    
//double at, it, ft, ot, statet, output_state
	//printf("Memcpy completed\n");
	dim3 n_threads(100,1,1);
	dim3 n_blocks(1,1,1);
	double result_0 = 0;
	double result_1 = 0;

	ExecuteLSTM<<<n_blocks,n_threads>>>(result_0, result_1, w_i_device, u_i_device, b_i_device, w_f_device, u_f_device, b_f_device, w_c_device, u_c_device, b_c_device, w_o_device, u_o_device, b_o_device, weights_device, bias_device, LSTM_results, at_0_device, at_1_device, it_0_device, it_1_device, ft_0_device, ft_1_device, ot_0_device, ot_1_device, statet_0_device, statet_1_device, u_0_device, u_1_device);
        hipDeviceSynchronize();
        printf("finishing executing forward propagation"); 
        //cudaThreadSynchronize();
        bpLSTM<<<n_blocks,n_threads>>>(result_0, result_1, w_i_device, u_i_device, b_i_device, w_f_device, u_f_device, b_f_device, w_c_device, u_c_device, b_c_device, w_o_device, u_o_device, b_o_device, weights_device, bias_device, LSTM_results, at_0_device, at_1_device, it_0_device, it_1_device, ft_0_device, ft_1_device, ot_0_device, ot_1_device, statet_0_device, statet_1_device, u_0_device, u_1_device);
        hipDeviceSynchronize();
        //cudaThreadSynchronize();
        printf("finishing executing back propagation");
}
